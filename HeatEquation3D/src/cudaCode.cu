
#include <hip/hip_runtime.h>




__global__  void computeT(double*** TBegin, double*** TEnd, int numX, int numY, int numZ, double Dx, double Dy, double Dz) {

    for (unsigned i = 1; i < numX - 1; ++i)
        for (unsigned j = 1; j < numY - 1; ++j)
            for (unsigned k = 1; k < numZ - 1; ++k) {
                TEnd[i][j][k] = TBegin[i][j][k] +
                    Dx * (TBegin[i + 1][j][k] - 2.0 * TBegin[i][j][k] + TBegin[i - 1][j][k]) +
                    Dy * (TBegin[i][j + 1][k] - 2.0 * TBegin[i][j][k] + TBegin[i][j - 1][k]) +
                    Dz * (TBegin[i][j][k + 1] - 2.0 * TBegin[i][j][k] + TBegin[i][j][k - 1]);
            }


}

